#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cmath>


#include <coreutils/classes/matrixes/Tensor.cuh>

#include <coreutils/functions/debug/print.hpp>
#include <coreutils/functions/sort/sortHelpers.hpp>
#include <coreutils/functions/math/simpleMath.hpp>

#include <coreutils/util/time.hpp>
#include <coreutils/util/cudaErrors.cuh>

using namespace coreutils::functions;
using namespace coreutils::classes::matrixes;

int Tensor::getLength () {
	return this->length;
}

int Tensor::getWidth () {
	return this->width;
}

int Tensor::getHeight () {
	return this->height;
}

float* Tensor::getArr () {
	return arr;
}

long long Tensor::getSize () {
	return memorySize;
}

float* Tensor::getData (int length, int width, int height) {
	if (this->length <= length || this->width <= width || this->height <= height) {
		std::cout << "Invalid input at getData";
		return nullptr;
	}
	return &this->arr[getIndex(length, width, height)];
}

int Tensor::getIndex (int l, int w, int h) const {
	return l * this->width * this->height + w * this->height + h;
}

void Tensor::shuffleEvery () {
	srand(GetTimeStamp().tv_sec + GetTimeStamp().tv_usec);
	for (int length = 0; length < this->length; length++) {
		for (int width = 0; width < this->width; width++) {
			for (int height = 0; height < this->height; height++) {
				sort::swap (&this->arr[getIndex(length, width, height)], 
								&this->arr[getIndex((double) rand() / RAND_MAX * length, (double) rand() / RAND_MAX * width, (double) rand() / RAND_MAX * height)]);

			}
		}
	}
}


int* Tensor::shuffleGroups () {
	int* order = new int[this->length];
	for (int length = 0; length < this->length; length++) {
		srand(GetTimeStamp().tv_sec + GetTimeStamp().tv_usec);
		double randomLength = rand() / RAND_MAX * length;
		order[length] = randomLength;
		for (int width = 0; width < this->width; width++) {
			for (int height = 0; height < this->height; height++) {
				sort::swap (&this->arr [getIndex(length, width, height)], 
								&this->arr [getIndex(randomLength, width, height)]);

			}
		}
	}
	return order;
}

void Tensor::shuffleGroups (int* order) {
	for (int length = 0; length < this->length; length++) {
		for (int width = 0; width < this->width; width++) {
			for (int height = 0; height < this->height; height++) {
				sort::swap (&this->arr [getIndex(length, width, height)], 
								&this->arr [getIndex(order[length], width, height)]);

			}
		}
	}
}

void Tensor::operator += (const Tensor* m2) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->arr [getIndex(i, j, k)] += m2->arr [getIndex(i, j, k)];
			}
		}
	}
}

void Tensor::operator -= (const Tensor* m2) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->arr [getIndex(i, j, k)] -= m2->arr [getIndex(i, j, k)];
			}
		}
	}
}

Tensor* Tensor::operator + (const Tensor* m2) {
	Tensor* M3D = new Tensor (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] + m2->arr [getIndex(i, j, k)];
			}
		}
	}

	return M3D;
}

Tensor* Tensor::operator - (const Tensor* m2) {
	Tensor* M3D = new Tensor (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] - m2->arr [getIndex(i, j, k)];
			}
		}
	}

	return M3D;
}

Tensor* Tensor::operator * (const Tensor* m2) {
	Tensor* M3D = new Tensor(this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] * m2->arr [getIndex(i, j, k)];
			}
		}
	}

	return M3D;
}

Tensor* Tensor::operator * (const float x) {
	Tensor* M3D = new Tensor (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i, j, k)] = this->arr [getIndex(i, j, k)] * x;
			}
		}
	}
	
	return M3D;
}

Tensor* Tensor::operator / (const Tensor* m2) {
	Tensor* M3D = new Tensor (this->length, this->width, this->height);

	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				M3D->arr[getIndex(i,j,k)] = this->arr [getIndex(i,j,k)] / m2->arr [getIndex(i,j,k)];
			}
		}
	}
	
	return M3D;
}

bool Tensor::equals (const Tensor* m2, double tolerance) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				if (m2->arr[getIndex(i, j, k)] - this->arr [getIndex(i, j, k)] > tolerance) {
					return false;
				}
			}
		}
	}

	return true;
}

void Tensor::randomize (double lowerBound, double upperBound) {
	double currentRandomNumber;
	srand(GetTimeStamp().tv_sec + GetTimeStamp().tv_usec);
	for (int i = 0; i < this->length; i++) {
		for (int j = 0; j < this->width; j++) {
			for (int k = 0; k < this->height; k++) {
				currentRandomNumber = ((double) rand()) / RAND_MAX * (upperBound - lowerBound) + lowerBound;
				this->arr [getIndex(i, j, k)] = currentRandomNumber;
			}
		}
	}
}

void Tensor::xavierRandomize (int l1, int w1, int h1, int l2, int w2, int h2) {
	double bound = sqrt(6) / (sqrt(l1 * w1 * h1 + l2 * w2 * h2));
	this->randomize(-bound, bound);
}

double Tensor::dotProduct (const Tensor* m2) {
	double output = 0;
	
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				output += this->arr [getIndex(i, j, k)] * m2->arr [getIndex(i, j, k)];
			}
		}
	}
	
	return output;
}

double Tensor::sum () {
	double output = 0;
	
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				output += this->arr [getIndex(i, j, k)];
			}
		}
	}
	
	return output;
}

void Tensor::insert (float data, int length, int width, int height) {
	this->arr[getIndex(length, width, height)] = data;
}

void Tensor::printMatrix () const {
	std::cout << '\n' << "{";
	for (int i = 0; i < this->length; i++) {
		std::cout << '\n' << "  {" << '\n';
		for (int j = 0; j < this->width; j++) {
			std::string out = "    {";
			for (int k = 0; k < this->height; k++) {
				out += std::to_string(this->arr [getIndex(i, j, k)]) + ", ";
			}
			out = out.substr(0, out.length () - 2);
			std::cout << out << "}" << '\n';
		}
		std::cout << "  }";
	}
	std::cout << '\n' << "}" << '\n';
}

void Tensor::setMatrix (Tensor* M3D) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->insert(*M3D->getData(i, j, k), i, j, k);
			}
		}
	}
}

void Tensor::setAll (double x) {
	for (int i = 0; i < length; i++) {
		for (int j = 0; j < width; j++) {
			for (int k = 0; k < height; k++) {
				this->insert(x, i, j, k);
			}
		}
	}
}

Tensor::Tensor (const int length, const int width, const int height) {
	this->length = length;
	this->width = width;
	this->height = height;
	gpuErrchk(hipHostMalloc((void **) &this->arr, length * width * height * sizeof(float)));
	this->memorySize = length * width * height * sizeof(float);
}

Tensor::Tensor (const Tensor &m3d) {
	this->length = m3d.length;
	this->width = m3d.width;
	this->height = m3d.height;
	gpuErrchk(hipHostMalloc((void **) &this->arr, length * width * height * sizeof(float)));
	gpuErrchk(hipMemcpy(this->arr, m3d.arr, this->length * this->width * this->height * sizeof(float), hipMemcpyHostToHost));
	this->memorySize = length * width * height * sizeof(float);
}

Tensor::Tensor () {
	this->length = 0;
	this->width = 0;
	this->height = 0;
	this->arr = nullptr;
	this->memorySize = 0;
}

Tensor::~Tensor () {
	hipHostFree(this->arr);
}